//--------------------------------------------*-C++-*---------------------------------------------//
/*!
 * \file   config/query_gpu.cu
 * \author Alex Long
 * \brief  Small CUDA code that prints the architecture version, used by CMake
 * \date   Thu Mat 21 15:53:51 2019
 * \note   Copyright (C) 2019-2023 Triad National Security, LLC., All rights reserved.
 *
 * This code is from wagonhelm.github.io/articles/2018-03/detecting-cuda-capability-with-cmake */
//------------------------------------------------------------------------------------------------//


#include <hip/hip_runtime.h>
#include <stdio.h>

// This ifdef block is related to the code found at src/device/device_gpu.h.  We need this code
// locally in this file because it is compled before cmake is done creating device/config.h.
#ifdef __HIP__
#include "hip/hip_runtime.h"
#define gpuDeviceProp hipDeviceProp_t
#define gpuGetDeviceProperties hipGetDeviceProperties
#define gpuGetLastError hipGetLastError
#define gpuSuccess hipSuccess
#define gpuGetErrorString hipGetErrorString
#define gpuError_t hipError_t
#else
#define gpuDeviceProp hipDeviceProp_t
#define gpuGetDeviceProperties hipGetDeviceProperties
#define gpuGetLastError hipGetLastError
#define gpuSuccess hipSuccess
#define gpuGetErrorString hipGetErrorString
#define gpuError_t hipError_t
#endif

int main(int argc, char **argv) {
  gpuDeviceProp dP;
  float min_cc = 3.0;

  int rc = gpuGetDeviceProperties(&dP, 0);
  if (rc != gpuSuccess) {
    gpuError_t error = gpuGetLastError();
    printf("CUDA/HIP error: %s", gpuGetErrorString(error));
    return rc; /* Failure */
  }
  if ((dP.major + (dP.minor / 10)) < min_cc) {
    printf("Min Compute Capability of %2.1f required:  %d.%d found\n Not building CUDA/HIP code",
           min_cc, dP.major, dP.minor);
    return 1; /* Failure */
  } else {
    // printf("-arch=sm_%d%d", dP.major, dP.minor);
    printf("%d%d", dP.major, dP.minor);
    return 0; /* Success */
  }
}

//------------------------------------------------------------------------------------------------//
// end of query_gpu.cu
//------------------------------------------------------------------------------------------------//
